#include "hip/hip_runtime.h"
#include "tile.h"

#define WINGDIAPI
#define APIENTRY
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

constexpr int BLOCK_SIZE = 16;
const dim3 THREADS{ BLOCK_SIZE, BLOCK_SIZE };
const dim3 GRID{ (GRID_WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE, (GRID_HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE };

class TileGrid
{
public:
	__host__ __device__ Tile* operator()(unsigned int x, unsigned int y)
	{
		if (x >= GRID_WIDTH || y >= GRID_HEIGHT)
		{
			return nullptr;
		}
		else
		{
			return &tiles[x + y * GRID_WIDTH];
		}
	}

	unsigned int tick;

private:
	Tile tiles[GRID_WIDTH * GRID_HEIGHT];
};

__device__ void SwapTiles(Tile* a, Tile* b)
{
	Tile copy = *a;
	*a = *b;
	*b = copy;
}

__device__ void SwapTiles(Tile& a, Tile& b)
{
	Tile copy = a;
	a = b;
	b = copy;
}

typedef void (*TileUpdate)(TileGrid&, unsigned int, unsigned int);
__device__ void AirUpdate(TileGrid& grid, unsigned int x, unsigned int y) { }
__device__ void SandUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	{
		Tile* below = grid(x, y - 1);

		if (below && below->type == Tile::Air)
		{
			SwapTiles(grid(x, y), below);

			return;
		}
	}

	{
		Tile* downLeft = grid(x - 1, y - 1);

		if (downLeft && downLeft->type == Tile::Air)
		{
			SwapTiles(grid(x, y), downLeft);

			return;
		}
	}

	{
		Tile* downRight = grid(x + 1, y - 1);

		if (downRight && downRight->type == Tile::Air)
		{
			SwapTiles(grid(x, y), downRight);

			return;
		}
	}
}

__device__ void WaterUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	Tile* below = grid(x, y - 1);

	if (below && below->type == Tile::Air)
	{
		SwapTiles(grid(x, y), below);

		return;
	}
	else
	{
		if ((grid.tick + x) % 2 == 0)
		{
			Tile* left = grid(x - 1, y);

			if (left && left->type == Tile::Air)
			{
				SwapTiles(grid(x, y), left);
			}
		}
		else if ((grid.tick + x) % 2 == 1)
		{
			Tile* right = grid(x + 1, y);

			if (right && right->type == Tile::Air)
			{
				SwapTiles(grid(x, y), right);
			}
		}
	}
}

__constant__ TileUpdate tileUpdateFns[Tile::TypeCount] =
{
	AirUpdate,
	SandUpdate,
	WaterUpdate
};

static TileGrid* grid = nullptr;
static hipGraphicsResource_t resource;
static hipArray_t array;
static hipSurfaceObject_t surface;

__global__ void _Update(TileGrid* grid)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < GRID_WIDTH && y < GRID_HEIGHT)
	{
		tileUpdateFns[(*grid)(x, y)->type](*grid, x, y);
	}
}

__global__ void _Render(TileGrid* grid, hipSurfaceObject_t surface)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < GRID_WIDTH && y < GRID_HEIGHT)
	{
		surf2Dwrite((*grid)(x, y)->Color(), surface, x * sizeof(uint32_t), y);
	}
}

void SetupGrid(unsigned int textureID)
{
	if (!grid)
	{
		hipMallocManaged((void**)&grid, sizeof(TileGrid));
		hipGraphicsGLRegisterImage(&resource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		hipGraphicsMapResources(1, &resource);
		hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0);
		hipGraphicsUnmapResources(1, &resource);

		hipResourceDesc desc;
		desc.resType = hipResourceTypeArray;
		desc.res.array.array = array;

		hipCreateSurfaceObject(&surface, &desc);
	}
}

void DestroyGrid()
{
	hipFree(grid);
	hipDestroySurfaceObject(surface);
	grid = nullptr;
}

void Set(unsigned int x, unsigned int y, Tile t)
{
	*(*grid)(x, y) = t;
}

void Update()
{
	_Update<<<GRID, THREADS>>>(grid);
	_Render<<<GRID, THREADS>>>(grid, surface);
	hipDeviceSynchronize();
	grid->tick++;
}