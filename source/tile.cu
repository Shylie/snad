#include "hip/hip_runtime.h"
#include "tile.h"

#ifndef WINGDIAPI
#define WINGDIAPI
#endif
#ifndef APIENTRY
#define APIENTRY
#endif
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

constexpr unsigned int MAX_TILE_MODIFY_DISTANCE_X = 2;
constexpr unsigned int MAX_TILE_MODIFY_DISTANCE_Y = 1;
constexpr unsigned int MAX_OFFSET_X = MAX_TILE_MODIFY_DISTANCE_X * 2 + 1;
constexpr unsigned int MAX_OFFSET_Y = MAX_TILE_MODIFY_DISTANCE_Y * 2 + 1;
constexpr int BLOCK_SIZE = 32;
const dim3 THREADS{ BLOCK_SIZE, BLOCK_SIZE };
const dim3 BASE_GRID{ (GRID_WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE, (GRID_HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE };
const dim3 UPDATE_GRID{ (BASE_GRID.x + MAX_OFFSET_X - 1) / MAX_OFFSET_X, (BASE_GRID.y + MAX_OFFSET_Y - 1) / MAX_OFFSET_Y };

struct XORRand
{
public:
	__host__ __device__ uint32_t operator()()
	{
		state ^= state << 13;
		state ^= state >> 17;
		state ^= state << 5;

		return state;
	}

	__host__ __device__ void SetState(uint32_t state)
	{
		this->state = state;
	}

private:
	uint32_t state;
};

class TileGrid
{
public:
	__host__ __device__ Tile* __restrict__ Tile(unsigned int x, unsigned int y)
	{
		if ((x >= GRID_WIDTH) | (y >= GRID_HEIGHT))
		{
			return nullptr;
		}
		else
		{
			return &tiles[x + y * GRID_WIDTH];
		}
	}

	__host__ __device__ XORRand& Random(unsigned int x, unsigned int y)
	{
		return rands[x + y * GRID_WIDTH];
	}

	__host__ __device__ unsigned int GetTick()
	{
		return tick;
	}

	__host__ void SetTick(unsigned int t)
	{
		tick = t;
	}

private:
	::Tile tiles[GRID_WIDTH * GRID_HEIGHT];
	XORRand rands[GRID_WIDTH * GRID_HEIGHT];
	unsigned int tick;
};

__device__ void SwapTiles(Tile* __restrict__ a, Tile* __restrict__ b)
{
	Tile copy = *a;
	*a = *b;
	*b = copy;
}

//__device__ void Move(TileGrid& grid, unsigned int x, unsigned int y, int dx, int dy)
//{
//	__builtin_assume(x < GRID_WIDTH);
//	__builtin_assume(y < GRID_HEIGHT);
//	__builtin_assume((dx < 0 ? -dx : dx) <= MAX_TILE_MODIFY_DISTANCE_X);
//	__builtin_assume((dy < 0 ? -dy : dy) <= MAX_TILE_MODIFY_DISTANCE_Y);
//}

typedef void (*TileUpdate)(TileGrid&, unsigned int, unsigned int);
__device__ void AirUpdate(TileGrid& grid, unsigned int x, unsigned int y) { }
__device__ void SandUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	Tile* __restrict__ me = grid.Tile(x, y);

	{
		Tile* __restrict__ below = grid.Tile(x, y - 1);

		if (below && below->Density() < me->Density())
		{
			SwapTiles(me, below);

			return;
		}
	}

	if (grid.Random(x, y)() % 2 == 0)
	{
		Tile* __restrict__ downLeft = grid.Tile(x - 1, y - 1);

		if (downLeft && downLeft->Density() < me->Density())
		{
			SwapTiles(me, downLeft);

			return;
		}
	}
	else
	{
		Tile* __restrict__ downRight = grid.Tile(x + 1, y - 1);

		if (downRight && downRight->Density() < me->Density())
		{
			SwapTiles(me, downRight);

			return;
		}
	}
}

__device__ void BasicLiquidUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	Tile* __restrict__ me = grid.Tile(x, y);
	Tile* __restrict__ below = grid.Tile(x, y - 1);

	if (below && below->Density() < me->Density())
	{
		SwapTiles(me, below);
	}
	else
	{
		unsigned int r = grid.Random(x, y)();
		Tile* moveTo;
		switch (r % 7)
		{
		case 0:
		case 1:
			moveTo = (grid.Tile(x - 1, y) && grid.Tile(x - 1, y)->Density() < me->Density()) ? grid.Tile(x - 2, y) : nullptr;
			break;

		case 2:
			moveTo = grid.Tile(x - 1, y);
			break;

		case 3:
			moveTo = grid.Tile(x + 1, y);
			break;

		case 4:
		case 5:
			moveTo = (grid.Tile(x + 1, y) && grid.Tile(x + 1, y)->Density() < me->Density()) ? grid.Tile(x + 2, y) : nullptr;
			break;

		default:
			moveTo = nullptr;
			break;
		}

		if (moveTo && moveTo->Density() < me->Density())
		{
			SwapTiles(me, moveTo);
		}
	}
}

__device__ void WaterUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	const float temp = grid.Tile(x, y)->data.temperature;
	if (temp > 100.0f)
	{
		*grid.Tile(x, y) = Tile(Tile::Air).SetTemperature(temp);
	}
	else
	{
		BasicLiquidUpdate(grid, x, y);
	}
}

__device__ void LavaUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	const float temp = grid.Tile(x, y)->data.temperature;
	if (temp < 700.0f)
	{
		*grid.Tile(x, y) = Tile(Tile::Stone).SetTemperature(temp);
	}
	else
	{
		BasicLiquidUpdate(grid, x, y);
	}
}

__device__ void StoneUpdate(TileGrid& grid, unsigned int x, unsigned int y)
{
	const float temp = grid.Tile(x, y)->data.temperature;
	if (temp > 700.0f)
	{
		*grid.Tile(x, y) = Tile(Tile::Lava).SetTemperature(temp);
	}
	else
	{
		SandUpdate(grid, x, y);
	}
}

__constant__ TileUpdate tileUpdateFns[Tile::__TypeCount] =
{
	AirUpdate,
	SandUpdate,
	WaterUpdate,
	LavaUpdate,
	StoneUpdate
};

static TileGrid* grid = nullptr;
static hipGraphicsResource_t resource;
static hipArray_t array;
static hipSurfaceObject_t surface;

__global__ void _Update(TileGrid* grid, unsigned int ofx, unsigned int ofy)
{
	const unsigned int x = (blockDim.x * blockIdx.x + threadIdx.x) * MAX_OFFSET_X + ofx;
	const unsigned int y = (blockDim.y * blockIdx.y + threadIdx.y) * MAX_OFFSET_Y + ofy;

	Tile* __restrict__ t = grid->Tile(x, y);
	if (t && t->lastUpdated <= grid->GetTick())
	{
		t->lastUpdated = grid->GetTick() + 1;
		tileUpdateFns[t->type](*grid, x, y);
	}
}

__device__ float GetTemperature(TileGrid& grid, Tile* stiles, bool* sokay, unsigned int sx, unsigned int sy, unsigned int x, unsigned int y, float fallback)
{
	const unsigned int sid = sx + sy * blockDim.x;
	if (sx < 32 && sy < 32 && sokay[sid])
	{
		return stiles[sid].data.temperature;
	}
	else
	{
		const Tile* __restrict__ tile = grid.Tile(x, y);
		if (tile)
		{
			return tile->data.temperature;
		}
	}

	return fallback;
}

__global__ void _UpdateTemp(TileGrid* grid)
{
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	const unsigned int sx = threadIdx.x;
	const unsigned int sy = threadIdx.y;

	const unsigned int sid = sx + sy * blockDim.x;

	__shared__ Tile stiles[1024];
	__shared__ bool sokay[1024];

	Tile* __restrict__ tile = grid->Tile(x, y);
	if (tile)
	{
		stiles[sid] = *tile;
		sokay[sid] = true;
	}
	else
	{
		sokay[sid] = false;
	}

	__syncthreads();

	float lfluxx;
	float rfluxx;
	float lfluxy;
	float rfluxy;

	if (sokay[sid])
	{
		const float fallback = stiles[sid].data.temperature;
		const float conductivity = stiles[sid].ThermalConductivity();

		// approximate the temperature gradient
		const float ldTdx = fallback - GetTemperature(*grid, stiles, sokay, sx - 1, sy, x - 1, y, fallback);
		const float rdTdx = GetTemperature(*grid, stiles, sokay, sx + 1, sy, x + 1, y, fallback) - fallback;

		const float ldTdy = fallback - GetTemperature(*grid, stiles, sokay, sx, sy - 1, x, y - 1, fallback);
		const float rdTdy = GetTemperature(*grid, stiles, sokay, sx, sy + 1, x, y + 1, fallback) - fallback;

		// approximated temperature flux based on temperature gradient
		lfluxx = -conductivity * ldTdx;
		rfluxx = -conductivity * rdTdx;

		lfluxy = -conductivity * ldTdy;
		rfluxy = -conductivity * rdTdy;
	}

	__syncthreads();

	if (sokay[sid])
	{
		{
			Tile* __restrict__ left = grid->Tile(x - 1, y);
			if (left)
			{
				atomicAdd(&left->data.temperature, -lfluxx / left->SpecificHeat());
				atomicAdd(&tile->data.temperature, lfluxx / tile->SpecificHeat());
			}
		}
		{
			Tile* __restrict__ right = grid->Tile(x + 1, y);
			if (right)
			{
				atomicAdd(&right->data.temperature, rfluxx / right->SpecificHeat());
				atomicAdd(&tile->data.temperature, -rfluxx / tile->SpecificHeat());
			}
		}
		{
			Tile* __restrict__ down = grid->Tile(x, y - 1);
			if (down)
			{
				atomicAdd(&down->data.temperature, -lfluxy / down->SpecificHeat());
				atomicAdd(&tile->data.temperature, lfluxy / tile->SpecificHeat());
			}
		}
		{
			Tile* __restrict__ up = grid->Tile(x, y + 1);
			if (up)
			{
				atomicAdd(&up->data.temperature, rfluxy / up->SpecificHeat());
				atomicAdd(&tile->data.temperature, -rfluxy / tile->SpecificHeat());
			}
		}
	}
}

__global__ void _Render(TileGrid* grid, hipSurfaceObject_t surface)
{
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	Tile* tile = grid->Tile(x, y);
	if (tile)
	{
		surf2Dwrite(tile->Color(), surface, x * sizeof(uint32_t), y);
	}
}

__global__ void _Setup(TileGrid* grid)
{
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < GRID_WIDTH && y < GRID_HEIGHT)
	{
		*grid->Tile(x, y) = Tile(Tile::Air);
		grid->Tile(x, y)->lastUpdated = grid->GetTick();
		grid->Random(x, y).SetState(x + y * GRID_WIDTH);
	}
}

void SetupGrid(unsigned int textureID)
{
	if (!grid)
	{
		hipMallocManaged((void**)&grid, sizeof(TileGrid));
		hipGraphicsGLRegisterImage(&resource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		hipGraphicsMapResources(1, &resource);
		hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0);
		hipGraphicsUnmapResources(1, &resource);

		hipResourceDesc desc;
		desc.resType = hipResourceTypeArray;
		desc.res.array.array = array;

		hipCreateSurfaceObject(&surface, &desc);

		grid->SetTick(0);

		_Setup<<<BASE_GRID, THREADS>>>(grid);
		hipDeviceSynchronize();
	}
}

void DestroyGrid()
{
	hipFree(grid);
	hipDestroySurfaceObject(surface);
	grid = nullptr;
}

void SetTile(unsigned int x, unsigned int y, Tile t)
{
	*grid->Tile(x, y) = t;
}

Tile GetTile(unsigned int x, unsigned int y)
{
	return *grid->Tile(x, y);
}

void Update()
{
	_UpdateTemp<<<BASE_GRID, THREADS>>>(grid);
	hipDeviceSynchronize();

	for (unsigned int ofx = 0; ofx < MAX_OFFSET_X; ofx++)
	{
		for (unsigned int ofy = 0; ofy < MAX_OFFSET_Y; ofy++)
		{
			_Update<<<UPDATE_GRID, THREADS>>>(grid, ofx, ofy);
			hipDeviceSynchronize();
		}
	}

	grid->SetTick(grid->GetTick() + 1);
}

void Render()
{
	_Render<<<BASE_GRID, THREADS>>>(grid, surface);
	hipDeviceSynchronize();
}